#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <sys/time.h>
#include <random>
#include "fdtd.h"
const size_t nz = 59;//8;
const size_t ny = 53;//2;
const size_t nx = 51;//2;
const size_t na = 10000;
#define GET(z, y, x, nz, ny, nx) ((z)*((ny)*(nx))+(y)*(nx)+(x))
int main(const int argc, const char* argv[])
{
    auto rand = std::mt19937_64(19950918);
    FDTD<float, nz, ny, nx, 4> fdtd;
    float *p0 = new float[nz*ny*nx];
    float *p1 = new float[nz*ny*nx];
    float *vel = new float[nz*ny*nx];
    float *sum = new float[nz*ny*nx];
    size_t *addr = new size_t[na];
    size_t *gpu_addr = new size_t[na];
    for(size_t i=0;i<nz;i++)
        for(size_t j=0;j<ny;j++)
            for(size_t k=0;k<nx;k++)
            {
                p0[GET(i,j,k,nz,ny,nx)] = float(rand())/rand.max();
                p1[GET(i,j,k,nz,ny,nx)] = float(rand())/rand.max();
                vel[GET(i,j,k,nz,ny,nx)] = float(rand())/rand.max();
                sum[GET(i,j,k,nz,ny,nx)] = 0.0;
            }
    for(size_t i=0;i<na;i++)
    {
        size_t x = rand()%nx, y = rand()%ny, z = rand()%nz;
        addr[i] = GET(z,y,x,nz,ny,nx);
        gpu_addr[i] = fdtd.addrTrans(z,y,x);
    }
    float *cpu_p0 = new float[nz*ny*nx];
    float *cpu_p1 = new float[nz*ny*nx];
    float *cpu_sum = new float[nz*ny*nx];
    memcpy(cpu_p0, p0, sizeof(float)*nz*ny*nx);
    memcpy(cpu_p1, p1, sizeof(float)*nz*ny*nx);
    memcpy(cpu_sum, sum, sizeof(float)*nz*ny*nx);
    
    struct timeval start, end;
    gettimeofday(&start, NULL);
    for(size_t t=0;t<10;t++)
    {
        for(size_t i=4;i<nz-4;i++)
            for(size_t j=4;j<ny-4;j++)
                for(size_t k=4;k<nx-4;k++)
                    p0[GET(i,j,k,nz,ny,nx)] = vel[GET(i,j,k,nz,ny,nx)]*1./(6*8+1)*(p1[GET(i,j,k,nz,ny,nx)]+
                                                     1.0*(p1[GET(i,j,k+1,nz,ny,nx)]+p1[GET(i,j,k-1,nz,ny,nx)])+
                                                     1.0*(p1[GET(i,j,k+2,nz,ny,nx)]+p1[GET(i,j,k-2,nz,ny,nx)])+
                                                     1.0*(p1[GET(i,j,k+3,nz,ny,nx)]+p1[GET(i,j,k-3,nz,ny,nx)])+
                                                     1.0*(p1[GET(i,j,k+4,nz,ny,nx)]+p1[GET(i,j,k-4,nz,ny,nx)])+
                                                     2.0*(p1[GET(i,j+1,k,nz,ny,nx)]+p1[GET(i,j-1,k,nz,ny,nx)])+
                                                     2.0*(p1[GET(i,j+2,k,nz,ny,nx)]+p1[GET(i,j-2,k,nz,ny,nx)])+
                                                     2.0*(p1[GET(i,j+3,k,nz,ny,nx)]+p1[GET(i,j-3,k,nz,ny,nx)])+
                                                     2.0*(p1[GET(i,j+4,k,nz,ny,nx)]+p1[GET(i,j-4,k,nz,ny,nx)])+
                                                     3.0*(p1[GET(i+1,j,k,nz,ny,nx)]+p1[GET(i-1,j,k,nz,ny,nx)])+
                                                     3.0*(p1[GET(i+2,j,k,nz,ny,nx)]+p1[GET(i-2,j,k,nz,ny,nx)])+
                                                     3.0*(p1[GET(i+3,j,k,nz,ny,nx)]+p1[GET(i-3,j,k,nz,ny,nx)])+
                                                     3.0*(p1[GET(i+4,j,k,nz,ny,nx)]+p1[GET(i-4,j,k,nz,ny,nx)]))
                                                     -p0[GET(i,j,k,nz,ny,nx)];
        for(size_t i=0;i<na;i++)p0[addr[i]] += 1;
        for(size_t i=0;i<nz;i++)
            for(size_t j=0;j<ny;j++)
                for(size_t k=0;k<nx;k++)
                    if(i == nz/2 || j == ny/3 || k == nx/4)
                        p0[GET(i,j,k,nz,ny,nx)] *= 0.9;
        for(size_t i=4;i<nz-4;i++)
            for(size_t j=4;j<ny-4;j++)
                for(size_t k=4;k<nx-4;k++)
                    sum[GET(i,j,k,nz,ny,nx)] += p0[GET(i,j,k,nz,ny,nx)]*p1[GET(i,j,k,nz,ny,nx)];
        printf("loop %lu\n", t);
        std::swap(p0, p1);
    }
    gettimeofday(&end, NULL);
    printf("CPU time %.6lf\n", double(end.tv_sec-start.tv_sec)+1e-6*double(end.tv_usec-start.tv_usec));
    fdtd.mallocCube("p0");
    fdtd.mallocCube("p1");
    fdtd.mallocCube("sum");
    fdtd.mallocCube("vel");
    fdtd.malloc<size_t>("addr", na);
    fdtd.transferCubeToGPU("p0", cpu_p0);
    fdtd.transferCubeToGPU("p1", cpu_p1);
    fdtd.transferCubeToGPU("sum", cpu_sum);
    fdtd.transferCubeToGPU("vel", vel);
    fdtd.transferToGPU("addr", gpu_addr, na);
    auto prop_kernel = [=] __device__ (size_t z, size_t y, size_t x, size_t addr, float *output, float* zl, float *yl, float *xl, float*vel, float scal)
    {
        output[addr] = vel[addr]*scal*(zl[0]+
                       1.0*(xl[1]+xl[-1])+
                       1.0*(xl[2]+xl[-2])+
                       1.0*(xl[3]+xl[-3])+
                       1.0*(xl[4]+xl[-4])+
                       2.0*(yl[1]+yl[-1])+
                       2.0*(yl[2]+yl[-2])+
                       2.0*(yl[3]+yl[-3])+
                       2.0*(yl[4]+yl[-4])+
                       3.0*(zl[1]+zl[-1])+
                       3.0*(zl[2]+zl[-2])+
                       3.0*(zl[3]+zl[-3])+
                       3.0*(zl[4]+zl[-4]))
                       -output[addr];
    };
    auto inject_kernel = [=] __device__ (size_t i, float *output, size_t *addr, float add)
    {
        if(addr[i] != -1) atomicAdd(&output[addr[i]], add);
    };
    auto filt_kernel = [=] __device__ (size_t z, size_t y, size_t x, size_t addr, float *output)
    {
        if(z == nz/2 || y == ny/3 || x == nx/4)
            output[addr] *= 0.9;
    };
    auto mul_kernel = [=] __device__ (size_t z, size_t y, size_t x, size_t addr, float *sum, float *p0, float *p1)
    {
        if(z >= 4 && y >= 4 && x >= 4 && z < nz-4 && y < ny-4 && x < nx-4)
            sum[addr] += p0[addr]*p1[addr];
    };
    gettimeofday(&start, NULL);
    std::string s0 = "p0", s1 = "p1";
    for(size_t t=0;t<10;t++)
    {
        fdtd.propagate(s0, s1, true, prop_kernel, fdtd.getCube("vel"), 1.0/(6*8+1));
        fdtd.inject(na, inject_kernel, fdtd.getCube(s0), fdtd.get<size_t>("addr"), 1.0);
        fdtd.filt(filt_kernel, fdtd.getCube(s0));
        fdtd.filt(mul_kernel, fdtd.getCube("sum"), fdtd.getCube(s0), fdtd.getCube(s1));
        fdtd.commCubeHalo(s0);
        fdtd.commCubeHalo(s1);
        fdtd.sync();
        std::swap(s0, s1);
    }
    gettimeofday(&end, NULL);
    printf("GPU time %.6lf\n", double(end.tv_sec-start.tv_sec)+1e-6*double(end.tv_usec-start.tv_usec));
    memset(cpu_p0, 0, sizeof(float)*nz*ny*nx);
    memset(cpu_p1, 0, sizeof(float)*nz*ny*nx);
    memset(cpu_sum, 0, sizeof(float)*nz*ny*nx);
    fdtd.transferCubeToCPU(cpu_p0, s0);
    fdtd.transferCubeToCPU(cpu_p1, s1);
    fdtd.transferCubeToCPU(cpu_sum, "sum");

    auto rank = MPI::COMM_WORLD.Get_rank();
    for(size_t i=0;i<nz;i++)
        for(size_t j=0;j<ny;j++)
            for(size_t k=0;k<nx;k++)
            {
                if(fabs(p0[GET(i,j,k,nz,ny,nx)]-cpu_p0[GET(i,j,k,nz,ny,nx)])>1e-5 ||
                   fabs(p1[GET(i,j,k,nz,ny,nx)]-cpu_p1[GET(i,j,k,nz,ny,nx)])>1e-5 ||
                   fabs(sum[GET(i,j,k,nz,ny,nx)]-cpu_sum[GET(i,j,k,nz,ny,nx)])>1e-5)
                    fprintf(stderr, "rank = %d, [%lu][%lu][%lu]: %lf %lf, %lf %lf, %lf %lf\n", rank, i, j, k, p0[GET(i,j,k,nz,ny,nx)], cpu_p0[GET(i,j,k,nz,ny,nx)], p1[GET(i,j,k,nz,ny,nx)], cpu_p1[GET(i,j,k,nz,ny,nx)], sum[GET(i,j,k,nz,ny,nx)], cpu_sum[GET(i,j,k,nz,ny,nx)]);
            }
    hipDeviceSynchronize();
}
