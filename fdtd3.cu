#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <chrono>
#include <random>
#include "stencil.h"
const size_t nz = 512;
const size_t ny = 512;
const size_t nx = 512;
const size_t na = 10000;
#define GET(z, y, x, nz, ny, nx) ((z)*((ny)*(nx))+(y)*(nx)+(x))
typedef uint32_t gpu_size_t;
typedef int32_t gpu_signed_size_t;
int main(const int argc, const char* argv[])
{
    Stencil<float, nz, ny, nx, 3, true, gpu_size_t, gpu_signed_size_t> stencil;
    float *p0 = new float[nz*ny*nx];
    float *p1 = new float[nz*ny*nx];

    #pragma omp parallel for
    for(size_t i=0;i<nz;i++)
    {
        auto rand = std::mt19937_64(19950918+i);
        for(size_t j=0;j<ny;j++)
            for(size_t k=0;k<nx;k++)
            {
                p0[GET(i,j,k,nz,ny,nx)] = float(rand())/rand.max();
                p1[GET(i,j,k,nz,ny,nx)] = float(rand())/rand.max();
            }
    }
    float *gpu_p0 = new float[nz*ny*nx];
    float *gpu_p1 = new float[nz*ny*nx];
    memcpy(gpu_p0, p0, sizeof(float)*nz*ny*nx);
    memcpy(gpu_p1, p1, sizeof(float)*nz*ny*nx);
    
    std::chrono::time_point<std::chrono::system_clock> start, end;
#ifndef SKIP_CPU
    start = std::chrono::system_clock::now();
    for(size_t t=0;t<1000;t++)
    {
        #pragma omp parallel for
        for(size_t i=3;i<nz-3;i++)
            for(size_t j=3;j<ny-3;j++)
                for(size_t k=3;k<nx-3;k++)
                    p0[GET(i,j,k,nz,ny,nx)] = (0.01f*p1[GET(i,j,k,nz,ny,nx)]+
                                              (0.02f*p1[GET(i,j,k+1,nz,ny,nx)]+0.03f*p1[GET(i,j,k-1,nz,ny,nx)])+
                                              (0.04f*p1[GET(i,j+1,k,nz,ny,nx)]+0.05f*p1[GET(i,j-1,k,nz,ny,nx)])+
                                              (0.06f*p1[GET(i+1,j,k,nz,ny,nx)]+0.07f*p1[GET(i-1,j,k,nz,ny,nx)])+
                                              (0.02f*p1[GET(i,j,k+2,nz,ny,nx)]+0.03f*p1[GET(i,j,k-2,nz,ny,nx)])+
                                              (0.04f*p1[GET(i,j+2,k,nz,ny,nx)]+0.05f*p1[GET(i,j-2,k,nz,ny,nx)])+
                                              (0.06f*p1[GET(i+2,j,k,nz,ny,nx)]+0.07f*p1[GET(i-2,j,k,nz,ny,nx)])+
                                              (0.02f*p1[GET(i,j,k+3,nz,ny,nx)]+0.03f*p1[GET(i,j,k-3,nz,ny,nx)])+
                                              (0.04f*p1[GET(i,j+3,k,nz,ny,nx)]+0.05f*p1[GET(i,j-3,k,nz,ny,nx)])+
                                              (0.06f*p1[GET(i+3,j,k,nz,ny,nx)]+0.07f*p1[GET(i-3,j,k,nz,ny,nx)]));
        if(stencil.getRank() == 0) printf("loop %lu\n", t);
        std::swap(p0, p1);
    }
    end = std::chrono::system_clock::now();
    printf("CPU time %.6lfs\n", 1e-6*(std::chrono::time_point_cast<std::chrono::microseconds>(end)-std::chrono::time_point_cast<std::chrono::microseconds>(start)));
#endif

    stencil.mallocCube("p0", true);
    stencil.mallocCube("p1", true);
    stencil.transferCubeToGPU("p0", gpu_p0);
    stencil.transferCubeToGPU("p1", gpu_p1);
    auto prop_kernel = [=] __device__ (gpu_size_t z, gpu_size_t y, gpu_size_t x, gpu_size_t addr, float *output, float* zl, gpu_signed_size_t sz, float *yl, gpu_signed_size_t sy, float *xl, gpu_signed_size_t sx)
    {
        output[addr] = (0.01f*zl[0]+
                        0.02f*xl[sx]+0.03f*xl[-sx]+0.04f*yl[sy]+0.05f*yl[-sy]+0.06f*zl[sz]+0.07f*zl[-sz]+
                        0.02f*xl[2*sx]+0.03f*xl[-2*sx]+0.04f*yl[2*sy]+0.05f*yl[-2*sy]+0.06f*zl[2*sz]+0.07f*zl[-2*sz]+
                        0.02f*xl[3*sx]+0.03f*xl[-3*sx]+0.04f*yl[3*sy]+0.05f*yl[-3*sy]+0.06f*zl[3*sz]+0.07f*zl[-3*sz]);
    };
    stencil.barrier();
    start = std::chrono::system_clock::now();
    std::string s0 = "p0", s1 = "p1";
    for(size_t t=0;t<1000;t++)
    {
        stencil.backupCubeHaloBackup(s0);
        stencil.propagateHaloTopBackup(s0, s1, true, prop_kernel);
        stencil.propagateHaloButtomBackup(s0, s1, true, prop_kernel);
        stencil.sync();
        stencil.propagate(s0, s1, true, prop_kernel);
        stencil.commCubeHaloBackup(s0);
        stencil.sync();
        stencil.restoreCubeHaloBackup(s0);
        stencil.sync();
        std::swap(s0, s1);
    }
    end = std::chrono::system_clock::now();
    printf("GPU time %.6lfs\n", 1e-6*(std::chrono::time_point_cast<std::chrono::microseconds>(end)-std::chrono::time_point_cast<std::chrono::microseconds>(start)));
    stencil.transferCubeToCPU(gpu_p0, s0);
    stencil.transferCubeToCPU(gpu_p1, s1);
    size_t rank = stencil.getRank();
#ifndef SKIP_CPU
    #pragma omp parallel for
    for(size_t i=0;i<nz;i++)
        for(size_t j=0;j<ny;j++)
            for(size_t k=0;k<nx;k++)
            {
                float dp0 = fabs(p0[GET(i,j,k,nz,ny,nx)]-gpu_p0[GET(i,j,k,nz,ny,nx)]),
                      dp1 = fabs(p1[GET(i,j,k,nz,ny,nx)]-gpu_p1[GET(i,j,k,nz,ny,nx)]);

                if((dp0/fabs(gpu_p0[GET(i,j,k,nz,ny,nx)])>1e-2 && dp0>1e-3) ||
                   (dp1/fabs(gpu_p0[GET(i,j,k,nz,ny,nx)])>1e-2 && dp0>1e-3))
                    fprintf(stderr, "rank = %lu, [%lu][%lu][%lu]: %lf %lf, %lf %lf\n", rank, i, j, k, p0[GET(i,j,k,nz,ny,nx)], gpu_p0[GET(i,j,k,nz,ny,nx)], p1[GET(i,j,k,nz,ny,nx)], gpu_p1[GET(i,j,k,nz,ny,nx)]);
            }
#endif
    return 0;
}
