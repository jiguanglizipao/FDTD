#include<hip/hip_runtime.h>
#include<stdio.h>

#define REAL float
#define NX 512 
#define NY 512 
#define NZ 512 
#define T  1000

#define BX 32
#define BY 32
#define BZ 1
#define GZ 16

const float cc = 0.01;
const float ce = 0.02;
const float cw = 0.03;
const float cs = 0.04;
const float cn = 0.05;
const float ct = 0.06;
const float cb = 0.07;

#define dimT 3
#define kDEP 1

#define stencil(curT, curH, curTB)                         \
	if (threadIdx.x > 0 && threadIdx.x < blockDim.x-1 &&   \
		threadIdx.y > 0 && threadIdx.y < blockDim.y-1){    \
		if (global_i > 0 && global_i < nx-1 &&             \
			global_j > 0 && global_j < ny-1){              \
			cur_##curT##_##curH =                          \
				  ce*cur_plane[curTB][lyidx][lxidx+1]      \
				 +cw*cur_plane[curTB][lyidx][lxidx-1]      \
				 +cs*cur_plane[curTB][lyidx+1][lxidx]      \
				 +cn*cur_plane[curTB][lyidx-1][lxidx]      \
				 +ct*cur_##curTB##_2                       \
				 +cb*cur_##curTB##_0                       \
				 +cc*cur_##curTB##_1;                      \
		}else{                                             \
			cur_##curT##_##curH =                          \
				  cur_##curTB##_1;                         \
		}                                                  \
	}

#define stencil_only(curT, curH, curTB)                    \
		cur_##curT##_##curH =                          \
			  ce*(cur_plane[curTB][lyidx][lxidx+1])      \
			 +cw*(cur_plane[curTB][lyidx][lxidx-1])      \
			 +cs*(cur_plane[curTB][lyidx+1][lxidx])      \
			 +cn*(cur_plane[curTB][lyidx-1][lxidx])      \
			 +ct*(cur_##curTB##_2)                      \
			 +cb*(cur_##curTB##_0)                       \
			 +cc*(cur_##curTB##_1);                      
	    

#define write_global_copy(idx_k0, slice, temp_k)           \
		if (threadIdx.x >= dimT &&                         \
			threadIdx.x < blockDim.x-dimT &&               \
			threadIdx.y >= dimT &&                         \
			threadIdx.y < blockDim.y-dimT){                \
			A[idx_k0+temp_k*slice] =                       \
					B[idx_k0+temp_k*slice];                \
		}

#define load_shared_t0_extra(cur_idx)                      \
		if (lyidx == 1)                                    \
			cur_plane[0][0][lxidx] =                       \
				B[cur_idx-nx];                             \
		else if (lyidx == blockDim.y)                      \
			cur_plane[0][blockDim.y+1][lxidx] =            \
				B[cur_idx+nx];                             \
		if (lxidx == 1)                                    \
			cur_plane[0][lyidx][0] =                       \
				B[cur_idx-1];                              \
		else if (lxidx == blockDim.x)                      \
			cur_plane[0][lyidx][blockDim.x+1] =            \
				B[cur_idx+1];                     

#define write_global_cal(idx_k0, slice, temp_k)             \
		if (threadIdx.x >= dimT &&                          \
			threadIdx.x < blockDim.x-dimT &&                \
			threadIdx.y >= dimT &&                          \
			threadIdx.y < blockDim.y-dimT){                 \
			A[idx_k0+temp_k*slice] =                        \
					ce*(cur_plane[dimT-1][lyidx][lxidx+1])  \
				   +cw*(cur_plane[dimT-1][lyidx][lxidx-1])  \
				   +cs*(cur_plane[dimT-1][lyidx+1][lxidx])  \
				   +cn*(cur_plane[dimT-1][lyidx-1][lxidx])  \
				   +ct*(cur_2_2)                            \
				   +cb*(cur_2_0)                            \
				   +cc*(cur_2_1);                           \
		}

__global__ void baseline(REAL* A, REAL* B, int64_t nx, int64_t ny, int64_t nz)
{
	int64_t i = threadIdx.x + blockDim.x*blockIdx.x;
	int64_t j = threadIdx.y + blockDim.y*blockIdx.y;
	int64_t kb = nz/gridDim.z*blockIdx.z;
	int64_t slice = nx*ny;

	int64_t k = kb > 0? kb: 1;
	int64_t ke = (kb+nz/gridDim.z<nz-1)? kb+nz/gridDim.z : nz-1;
	int64_t idx = i + j*nx + k*slice;
	for (; k < ke; k++){
		if (i > 0 && i < nx && j > 0 && j < ny){
			A[idx] = ce*B[idx+1] + cw*B[idx-1] + cs*B[idx+nx] + cn*B[idx-nx]
					+ct*B[idx+slice] + cb*B[idx-slice] + cc*B[idx];
			idx += slice;
		}
	}

	return;
}

__global__ void temporal_blocking(REAL* A, REAL* B, int64_t nx, int64_t ny, int64_t nz)
{
	int64_t global_i = (threadIdx.x-dimT)
				 + (blockDim.x-2*dimT)*blockIdx.x;
	int64_t global_j = (threadIdx.y-dimT) 
				 + (blockDim.y-2*dimT)*blockIdx.y;
	int64_t slice = nx*ny;

	int64_t gidx = global_i + global_j*nx;
	int64_t lxidx = threadIdx.x;
	int64_t lyidx = threadIdx.y;

	/*
	REAL top[dimT][kDEP];
	REAL mid[dimT][1];
	REAL bot[dimT][kDEP];
	REAL cur[dimT][1];
	*/
	//REAL cur[dimT][2*kDEP+2];
	REAL cur_0_0, cur_0_1, cur_0_2, cur_0_3;//from bottom to up
	REAL cur_1_0, cur_1_1, cur_1_2, cur_1_3;
	REAL cur_2_0, cur_2_1, cur_2_2, cur_2_3;
	//int64_t cur_size = 2*kDEP+2;

	__shared__ REAL cur_plane[dimT][BY][BX];

	/////////////////////////////////////////////////////
	//phase1
	/////////////////////////////////////////////////////
	//s1

	if (global_i >= 0 && global_i < nx &&
		global_j >= 0 && global_j < ny){

		cur_0_0 = B[gidx]; 
		cur_1_0 = cur_0_0; cur_2_0 = cur_0_0;

		//if (threadIdx.x >= (dimT-1) && 
		//	threadIdx.x <= blockDim.x-dimT &&
		//	threadIdx.y >= (dimT-1) && 
		//	threadIdx.y <= blockDim.y-dimT){
		//	A[gidx] = B[gidx];
		//}
		write_global_copy(gidx, slice, 0)

		//s2 & s3
		cur_0_1 = B[gidx+slice]; cur_0_2 = B[gidx+2*slice];


		/////////////////////////////////
		//load s2 into cur_plane();
		cur_plane[0][lyidx][lxidx] = cur_0_1;
		//load_shared_t0_extra(gidx+slice)
		__syncthreads();

		/////////////////////////////////////////////////
		//s4~s13
		//s4
		cur_0_3 = B[gidx+3*slice];
		//s5
		stencil(1,1,0)

		//shared_memory update
		__syncthreads();
		cur_0_0 = cur_0_1; cur_0_1 = cur_0_2; cur_0_2 = cur_0_3;
		cur_plane[0][lyidx][lxidx] = cur_0_1;
		//load_shared_t0_extra(gidx+2*slice)
		__syncthreads();

		//s6
		cur_0_3 = B[gidx+4*slice];
		//s7
		stencil(1,2,0)

		//shared_memory update
		__syncthreads();
		cur_0_0 = cur_0_1; cur_0_1 = cur_0_2; cur_0_2 = cur_0_3;
		cur_plane[0][lyidx][lxidx] = cur_0_1;
		//load_shared_t0_extra(gidx+3*slice)
		cur_plane[1][lyidx][lxidx] = cur_1_1;
		__syncthreads();

		//s8
		cur_0_3 = B[gidx+5*slice];
		//s9
		stencil(1,3,0)
		//s10
		stencil(2,1,1)
		//shared memory update
		//update s6,s7
		__syncthreads();
		cur_0_0 = cur_0_1; cur_0_1 = cur_0_2; cur_0_2 = cur_0_3;
		cur_plane[0][lyidx][lxidx] = cur_0_1;
		//load_shared_t0_extra(gidx+4*slice)
		cur_1_0 = cur_1_1; cur_1_1 = cur_1_2; cur_1_2 = cur_1_3;
		cur_plane[1][lyidx][lxidx] = cur_1_1;
		__syncthreads();

		//s11
		cur_0_3 = B[gidx+6*slice];
		//s12
		stencil(1,3,0)
		//s13
		stencil(2,2,1)

		//shared memory update
		__syncthreads();
		//update s8
		cur_0_0 = cur_0_1; cur_0_1 = cur_0_2; cur_0_2 = cur_0_3;
		cur_plane[0][lyidx][lxidx] = cur_0_1;
		//load_shared_t0_extra(gidx+5*slice)
		//update s9
		cur_1_0 = cur_1_1; cur_1_1 = cur_1_2; cur_1_2 = cur_1_3;
		cur_plane[1][lyidx][lxidx] = cur_1_1;
		//update s10
		cur_plane[2][lyidx][lxidx] = cur_2_1;
		__syncthreads();



		////check s13
		//if (threadIdx.x >= (dimT-1) && 
		//	threadIdx.x <= blockDim.x-dimT &&
		//	threadIdx.y >= (dimT-1) && 
		//	threadIdx.y <= blockDim.y-dimT){

		//	for (int64_t temp_k = 2; temp_k <= 2 ; temp_k++){
		//		//A[gidx+temp_k*slice] = B[gidx+temp_k*slice];
		//		if (global_i > 0 && global_i < nx-1 &&
		//			global_j > 0 && global_j < ny-1){
		//		//A[gidx+temp_k*slice] = cur[2][temp_k%cur_size];
		//			A[gidx+temp_k*slice] = cur_2_2;
		//		}
		//	}
		//}

		/////////////////////////////////////////////////////
		//phase2
		/////////////////////////////////////////////////////
		//now focus on the index of t=dimT
		for (int64_t temp_k = 1; temp_k < nz-2*dimT; temp_k++){
			//load t=0...
			//for t=0, the k index of buffer loading is
			int64_t k_index = 2*dimT+temp_k;
			cur_0_3 = B[gidx+k_index*slice];

			if (threadIdx.x > 0 && threadIdx.x < blockDim.x-1 &&
				threadIdx.y > 0 && threadIdx.y < blockDim.y-1){
				if (global_i > 0 && global_i < nx-1 &&
					global_j > 0 && global_j < ny-1){
					stencil_only(1,3,0)
					stencil_only(2,3,1)
					write_global_cal(gidx,slice,temp_k)

				}else{
					cur_1_3 = cur_0_1;
					cur_2_3 = cur_1_1;
				}
			}

			//update shared-memory buffer=>cur_plane
			__syncthreads();
			cur_0_0 = cur_0_1; cur_0_1 = cur_0_2; cur_0_2 = cur_0_3;
			cur_plane[0][lyidx][lxidx] = cur_0_1;
			//load_shared_t0_extra(gidx+(k_index-1)*slice)

			cur_1_0 = cur_1_1; cur_1_1 = cur_1_2; cur_1_2 = cur_1_3;
			cur_plane[1][lyidx][lxidx] = cur_1_1;
			cur_2_0 = cur_2_1; cur_2_1 = cur_2_2; cur_2_2 = cur_2_3;
			cur_plane[2][lyidx][lxidx] = cur_2_1;
			__syncthreads();
		}

		/////////////////////////////////////////////////////
		//phase3
		/////////////////////////////////////////////////////
		int64_t temp_k = nz-6;
		if (threadIdx.x > 0 && threadIdx.x < blockDim.x-1 &&
			threadIdx.y > 0 && threadIdx.y < blockDim.y-1){
			if (global_i > 0 && global_i < nx-1 &&
				global_j > 0 && global_j < ny-1){
				stencil_only(1,3,0)
				stencil_only(2,3,1)
				write_global_cal(gidx, slice, temp_k)
			}else{
				cur_1_3 = cur_0_1;	
				cur_2_3 = cur_1_1;
			}
		}

		__syncthreads();
		//cur_0_0 = cur_0_1; cur_0_1 = cur_0_2; cur_0_2 = cur_0_3;
		//cur_plane[0][lyidx][lxidx] = cur_0_1;
		cur_1_0 = cur_1_1; cur_1_1 = cur_1_2; cur_1_2 = cur_1_3;
		cur_plane[1][lyidx][lxidx] = cur_1_1;
		cur_2_0 = cur_2_1; cur_2_1 = cur_2_2; cur_2_2 = cur_2_3;
		cur_plane[2][lyidx][lxidx] = cur_2_1;
		__syncthreads();

		temp_k = nz-5;
		if (threadIdx.x > 0 && threadIdx.x < blockDim.x-1 &&
			threadIdx.y > 0 && threadIdx.y < blockDim.y-1){
			if (global_i > 0 && global_i < nx-1 &&
				global_j > 0 && global_j < ny-1){
				cur_1_3 = cur_0_2;
				stencil_only(2,3,1)
				write_global_cal(gidx,slice,temp_k)
			}else{
				cur_1_3 = cur_0_1;	
				cur_2_3 = cur_1_1;
			}
		}

		__syncthreads();
		//cur_0_0 = cur_0_1; cur_0_1 = cur_0_2; cur_0_2 = cur_0_3;
		//cur_plane[0][lyidx][lxidx] = cur_0_1;
		cur_1_0 = cur_1_1; cur_1_1 = cur_1_2; cur_1_2 = cur_1_3;
		cur_plane[1][lyidx][lxidx] = cur_1_1;
		cur_2_0 = cur_2_1; cur_2_1 = cur_2_2; cur_2_2 = cur_2_3;
		cur_plane[2][lyidx][lxidx] = cur_2_1;
		__syncthreads();

		temp_k = nz-4;
		if (threadIdx.x > 0 && threadIdx.x < blockDim.x-1 &&
			threadIdx.y > 0 && threadIdx.y < blockDim.y-1){
			if (global_i > 0 && global_i < nx-1 &&
				global_j > 0 && global_j < ny-1){
				stencil_only(2,3,1)
				write_global_cal(gidx,slice,temp_k)
			}else{
				cur_1_3 = cur_0_1;	
				cur_2_3 = cur_1_1;
			}
		}

		__syncthreads();
		//cur_0_0 = cur_0_1; cur_0_1 = cur_0_2; cur_0_2 = cur_0_3;
		//cur_plane[0][lyidx][lxidx] = cur_0_1;
		//cur_1_0 = cur_1_1; cur_1_1 = cur_1_2; cur_1_2 = cur_1_3;
		//cur_plane[1][lyidx][lxidx] = cur_1_1;
		cur_2_0 = cur_2_1; cur_2_1 = cur_2_2; cur_2_2 = cur_2_3;
		cur_plane[2][lyidx][lxidx] = cur_2_1;
		__syncthreads();

		temp_k = nz-3;
		if (global_i > 0 && global_i < nx-1 &&
			global_j > 0 && global_j < ny-1){
			cur_2_3 = cur_1_2;
			write_global_cal(gidx,slice,temp_k)
		}else{
			cur_1_3 = cur_0_1;	
			cur_2_3 = cur_1_1;
		}

		__syncthreads();
		//cur_0_0 = cur_0_1; cur_0_1 = cur_0_2; cur_0_2 = cur_0_3;
		//cur_plane[0][lyidx][lxidx] = cur_0_1;
		//cur_1_0 = cur_1_1; cur_1_1 = cur_1_2; cur_1_2 = cur_1_3;
		//cur_plane[1][lyidx][lxidx] = cur_1_1;
		cur_2_0 = cur_2_1; cur_2_1 = cur_2_2; cur_2_2 = cur_2_3;
		cur_plane[2][lyidx][lxidx] = cur_2_1;
		__syncthreads();

		temp_k = nz-2;
		if (global_i > 0 && global_i < nx-1 &&
			global_j > 0 && global_j < ny-1){
			write_global_cal(gidx, slice, temp_k)
		}

		//__syncthreads();
		//cur_0_0 = cur_0_1; cur_0_1 = cur_0_2; cur_0_2 = cur_0_3;
		//cur_plane[0][lyidx][lxidx] = cur_0_1;
		//cur_1_0 = cur_1_1; cur_1_1 = cur_1_2; cur_1_2 = cur_1_3;
		//cur_plane[1][lyidx][lxidx] = cur_1_1;
		//cur_2_0 = cur_2_1; cur_2_1 = cur_2_2; cur_2_2 = cur_2_3;
		//cur_plane[2][lyidx][lxidx] = cur_2_1;
		//__syncthreads();

		temp_k = nz-1;
		if (global_i > 0 && global_i < nx-1 &&
			global_j > 0 && global_j < ny-1){
			write_global_copy(gidx,slice,temp_k)
		}
	}
	return;
}

//#define check
#define checkT dimT

int main(){

	int64_t size = sizeof(REAL)*NX*NY*NZ;
	REAL* host_A = (REAL*)malloc(size);
	REAL* host_B = (REAL*)malloc(size);
	REAL* host_RES = (REAL*)malloc(size);

	for (int64_t k = 0; k < NZ; k++)
		for (int64_t j = 0; j < NY; j++)
			for (int64_t i = 0; i < NX; i++){
				host_B[k*NY*NX+j*NX+i] = i - j + 1.0/(k+1);	
				host_A[k*NY*NX+j*NX+i] = i - j + 1.0/(k+1);	
			}

	//cudaSetDevice(2);
	REAL *dev_A, *dev_B;
	hipMalloc(&dev_A, size);
	hipMalloc(&dev_B, size);
	hipMemcpy(dev_B, host_B, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_A, host_B, size, hipMemcpyHostToDevice);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float elapsed_time;
	double flops;

	//dim3 threadPerBlock(BX, BY, BZ);
	//dim3 blockPerGrid((NX+BX-1)/BX, (NY+BY-1)/BY, GZ);

	///////////////////////////////////////////////////////////////
	//baseline
	hipEventRecord(start, 0);
	/*
	for (int64_t t = 0; t < T; t++){
		baseline<<<blockPerGrid, threadPerBlock>>>(dev_A, dev_B, NX, NY, NZ);		
		REAL* tmp = dev_B;
		dev_B = dev_A;
		dev_A = tmp;
	}
	*/
	dim3 tpb(BX, BY, BZ);
	dim3 bpg((NX+BX-2*dimT-1)/(BX-2*dimT), 
			 (NY+BY-2*dimT-1)/(BY-2*dimT), 1);
	for (int64_t t = 0; t < T; t += dimT){
		temporal_blocking<<<bpg, tpb>>>(dev_A, dev_B, NX, NY, NZ);
		REAL* tmp = dev_B;
		dev_B = dev_A;
		dev_A = tmp;
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipError_t err;
	if ((err=hipGetLastError()) != hipSuccess)
		printf("baseline: wrong: %s!!!\n", hipGetErrorString(err));
	hipEventElapsedTime(&elapsed_time, start, stop);

	printf("baseline: elapsed time = %f ms\n", elapsed_time);
	flops = 1.0*13*(NX-2)*(NY-2)*(NZ-2)*T/1.e+6;
	//flops = 1.0*13*NX*NY*NZ*T/1.e+6;
	flops /= elapsed_time;
	printf("baseline: Gflops = %lf\n", flops);
	///////////////////////////////////////////////////////////////


	///////////////////////////////////////////////////////////////
	//check result
#ifdef check
	cudaMemcpy(host_RES, dev_B, size, cudaMemcpyDeviceToHost);
	for (int64_t t = 0; t < T; t++){
		for (int64_t k = 1; k < NZ-1; k++)
			for (int64_t j = 1; j < NY-1; j++)
				for (int64_t i = 1; i < NX-1; i++)
					host_A[k*NY*NX+j*NX+i] = 
						ce*host_B[k*NY*NX+j*NX+i+1]
					   +cw*host_B[k*NY*NX+j*NX+i-1]
					   +cs*host_B[k*NY*NX+(j+1)*NX+i]
					   +cn*host_B[k*NY*NX+(j-1)*NX+i]
					   +ct*host_B[(k+1)*NY*NX+j*NX+i]
					   +cb*host_B[(k-1)*NY*NX+j*NX+i]
					   +cc*host_B[k*NY*NX+j*NX+i];

		REAL *tmp = host_A;
		host_A = host_B;
		host_B = tmp;
	}
	for (int64_t k = 0; k < NZ; k++)
		for (int64_t j = 0; j < NY; j++)
			for (int64_t i = 0; i < NX; i++)
				if (host_B[k*NY*NX+j*NX+i] != 
						host_RES[k*NY*NX+j*NX+i])
					printf("host_B[%d][%d][%d] = %f\t" 
						   "host_RES[%d][%d][%d] = %f\n", 
						   k, j, i, host_B[k*NY*NX+j*NX+i], 
						   k, j, i, host_RES[k*NY*NX+j*NX+i]);
#endif


	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	return 0;
}
